#include "hip/hip_runtime.h"
/* Copyright 2019 Authors. All Rights Reserved.

 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Author: Patrick Wieschollek, <mail@patwie.com>, 2019
 *
 */

#if __HIPCC__

#include <iostream>

#include "include/cuda_index.h"
#include "include/cuda_utils.h"

namespace {

struct AddSharedMemoryCUDAKernel : public cuda::Kernel {
  void Launch(hipStream_t stream = 0) override {
    dim3 block(2);
    dim3 grid(1);

    cuda::SharedMemory shm;
    shm.add<float>(5);
    shm.add<int>(3);

    cuda::Run<<<grid, block, shm.bytes, stream>>>(*this);
  }

  __device__ __forceinline__ void operator()() const override {
    cuda::SharedMemory shm;
    float* floats_5 = shm.ref<float>(5);
    int* ints_3 = shm.ref<int>(3);

    if (threadIdx.x == 0) {
      floats_5[0] = 1.f;
      floats_5[1] = 2.f;
      floats_5[2] = 3.f;
      floats_5[3] = 4.f;
      floats_5[4] = 5.f;

      ints_3[0] = 11;
      ints_3[1] = 22;
      ints_3[2] = 33;
    }
    __syncthreads();
    if (threadIdx.x == 1) {
      float float_sum = 0;
      for (int i = 0; i < 5; ++i) {
        float_sum += floats_5[i];
        floats_5[i] = 0;
      }
      int int_sum = 0;
      for (int i = 0; i < 3; ++i) {
        int_sum += ints_3[i];
        ints_3[i] = 0;
      }

      printf("float sum: %f\n", float_sum);
      printf("int sum: %d\n", int_sum);
    }
  }
};
}  // namespace

int main(int argc, char const* argv[]) {
  AddSharedMemoryCUDAKernel kernel;
  kernel.Launch();
  ASSERT_CUDA(hipDeviceSynchronize());
  return 0;
}

#endif  // __HIPCC__
